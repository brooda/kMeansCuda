#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "classic.h"
#include "parallel.h"

float randFloat(float a) {
    return ((float) rand() / (float) (RAND_MAX)) * a;
}

void usage()
{
    printf("Required 2 input parameter: file length and k (number of clusters)");
}

int main(int argc, char **argv) {
    if (argc != 3)
    {
        usage();
    }

    int cardinality = atoi(argv[1]);
    int k = atoi(argv[2]);

    char* filePath = new char[30];
    sprintf(filePath, "input/%d/%d.csv", k, cardinality);

    char* line = NULL;
    size_t len = 0;
    ssize_t read;

    FILE* fp = fopen(filePath, "r");

    float* xs = new float[cardinality];
    float* ys = new float[cardinality];
    float* zs = new float[cardinality];

    int i=0;
    while ((read = getline(&line, &len, fp)) != -1) {
        float x;
        float y;
        float z;

        sscanf(line, "%f,%f,%f", &x,&y,&z );

        xs[i] = x;
        ys[i] = y;
        zs[i++] = z;
    }

    fclose(fp);

    float* startCentroidX = new float[k];
    float* startCentroidY = new float[k];
    float* startCentroidZ = new float[k];

    srand(2);
    for (int i=0; i<k; i++)
    {
        startCentroidX[i] = randFloat(4);
        startCentroidY[i] = randFloat(4);
        startCentroidZ[i] = randFloat(4);
    }
    printf("\n");

    clock_t start, end;
    double cpu_time_used;


    int printCentroids = 0;

    start = clock();
    int* classes = kMeans(k, cardinality, xs, ys, zs, startCentroidX, startCentroidY, startCentroidZ, printCentroids);
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Sequential took %f seconds to execute \n", cpu_time_used);

    start = clock();
    int* classesParallel = kMeansParallel(k, cardinality, xs, ys, zs, startCentroidX, startCentroidY, startCentroidZ, printCentroids);
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Parallel took %f seconds to execute \n", cpu_time_used);

    int mismatched = 0;
    for(int i=0; i<cardinality; i++)
    {
        if (classes[i] != classesParallel[i])
        {
            mismatched++;
        }
    }
    printf("Number of mismatched: %d \n", mismatched);

    char* outputFile = new char[30];
    sprintf(outputFile, "results/%d.txt", cardinality);
    FILE *f = fopen(outputFile, "ab+");

    for (int i=0; i<cardinality - 1; i++)
    {
        fprintf(f,"%d,", classes[i]);
    }

    fprintf(f,"%d", classes[cardinality - 1]);
    fclose(f);

    return 0;
}
